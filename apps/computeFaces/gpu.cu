#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "umesh/UMesh.h"
#ifdef __HIPCC__
# include <thrust/sort.h>
#endif
# ifdef UMESH_HAVE_TBB
#  include "tbb/parallel_sort.h"
# endif
#include <set>
#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#endif
#include <algorithm>
#include <string.h>

#ifndef PRINT
#ifdef __CUDA_ARCH__
# define PRINT(va) /**/
# define PING /**/
#else
# define PRINT(var) std::cout << #var << "=" << var << std::endl;
#ifdef __WIN32__
# define PING std::cout << __FILE__ << "::" << __LINE__ << ": " << __FUNCTION__ << std::endl;
#else
# define PING std::cout << __FILE__ << "::" << __LINE__ << ": " << __PRETTY_FUNCTION__ << std::endl;
#endif
#endif
#endif

#define CUDA_CHECK( call )                                              \
  {                                                                     \
    hipError_t rc = call;                                              \
    if (rc != hipSuccess) {                                            \
      fprintf(stderr,                                                   \
              "CUDA call (%s) failed with code %d (line %d): %s\n",     \
              #call, rc, __LINE__, hipGetErrorString(rc));             \
      throw std::runtime_error("fatal cuda error");                     \
    }                                                                   \
  }

#define CUDA_CALL(call) CUDA_CHECK(cuda##call)

#define CUDA_CHECK2( where, call )                                      \
  {                                                                     \
    hipError_t rc = call;                                              \
    if(rc != hipSuccess) {                                             \
      if (where)                                                        \
        fprintf(stderr, "at %s: CUDA call (%s) "                        \
                "failed with code %d (line %d): %s\n",                  \
                where,#call, rc, __LINE__, hipGetErrorString(rc));     \
      fprintf(stderr,                                                   \
              "CUDA call (%s) failed with code %d (line %d): %s\n",     \
              #call, rc, __LINE__, hipGetErrorString(rc));             \
      throw std::runtime_error("fatal cuda error");                     \
    }                                                                   \
  }

#define CUDA_SYNC_CHECK()                                       \
  {                                                             \
    hipDeviceSynchronize();                                    \
    hipError_t rc = hipGetLastError();                        \
    if (rc != hipSuccess) {                                    \
      fprintf(stderr, "error (%s: line %d): %s\n",              \
              __FILE__, __LINE__, hipGetErrorString(rc));      \
      throw std::runtime_error("fatal cuda error");             \
    }                                                           \
  }

namespace umesh {
  
#ifndef __CUDA_ARCH__
  typedef vec4i int4;
  using std::swap;
#else
  template<typename T>
  inline __umesh_both__ void swap(T &a, T &b)
  {
    T c = a; a = b; b = c;
  }
  
#endif
  
  struct PrimFacetRef{
    /*! only 4 possible types (tet, pyr, wedge, or hex) */
    uint64_t primType:3;
    /*! only 8 possible facet it could be (in a hex) */
    uint64_t facetIdx:3;
    int64_t  primIdx:58;
  };
  
  struct UMESH_ALIGN(16) Facet {
    int4         vertexIdx;
    PrimFacetRef prim;
    int          orientation;
  };
  
  struct FacetComparator {
    inline __umesh_both__
    bool operator()(const Facet &a, const Facet &b) const {
      return
        (a.vertexIdx.x < b.vertexIdx.x)
        ||
        ((a.vertexIdx.x == b.vertexIdx.x) &&
         (a.vertexIdx.y <  b.vertexIdx.y))
        ||
        ((a.vertexIdx.x == b.vertexIdx.x) &&
         (a.vertexIdx.y == b.vertexIdx.y) &&
         (a.vertexIdx.z <  b.vertexIdx.z))
        ||
        ((a.vertexIdx.x == b.vertexIdx.x) &&
         (a.vertexIdx.y == b.vertexIdx.y) &&
         (a.vertexIdx.z == b.vertexIdx.z) &&
         (a.vertexIdx.w <  b.vertexIdx.w));
    }
  };


  struct SharedFace {
    int4 vertexIdx;
    PrimFacetRef onFront, onBack;
  };
               
  void usage(const std::string &error)
  {
    if (error == "") std::cout << "Error: " << error << "\n\n";
    std::cout << "Usage: ./umeshComputeFaces{CPU|GPU} in.umesh -o out.faces\n";
    exit(error != "");
  }

  /*! describes input data through plain pointers, so we can run the
    same algorithm once with std::vector::data() (on the host) or
    cuda-malloced data (on gpu) */
  struct InputMesh {
    Tet   *tets;
    size_t numTets;
    Pyr   *pyrs;
    size_t numPyrs;
    Wedge *wedges;
    size_t numWedges;
    Hex   *hexes;
    size_t numHexes;
  };

  // ==================================================================
  // compute vertex order stage
  // ==================================================================
  
  inline __umesh_both__
  void computeUniqueVertexOrder(Facet &facet)
  {
    int4 idx = facet.vertexIdx;
    if (idx.w < 0) {
      if (idx.y < idx.x)
        { swap(idx.x,idx.y); facet.orientation = 1-facet.orientation; }
      if (idx.z < idx.x)
        { swap(idx.x,idx.z); facet.orientation = 1-facet.orientation; }
      if (idx.z < idx.y)
        { swap(idx.y,idx.z); facet.orientation = 1-facet.orientation; }
    } else {
      int lv = idx.x, li=0;
      if (idx.y < lv) { lv = idx.y; li = 1; }
      if (idx.z < lv) { lv = idx.z; li = 2; }
      if (idx.w < lv) { lv = idx.w; li = 3; }

      switch (li) {
      case 0: idx = { idx.x,idx.y,idx.z,idx.w }; break;
      case 1: idx = { idx.y,idx.z,idx.w,idx.x }; break;
      case 2: idx = { idx.z,idx.w,idx.x,idx.y }; break;
      case 3: idx = { idx.w,idx.x,idx.y,idx.z }; break;
      };

      if (idx.w < idx.y) {
        facet.orientation = 1-facet.orientation;
        swap(idx.w,idx.y);
      }
    }
    facet.vertexIdx = idx;
  }

#ifdef __HIPCC__
  __global__
  void computeUniqueVertexOrderLaunch(Facet *facets, size_t numFacets)
  {
    size_t jobIdx = size_t(blockIdx.x)*blockDim.x+threadIdx.x;
    if (jobIdx >= numFacets)
      return;
    computeUniqueVertexOrder(facets[jobIdx]);
  }

  void computeUniqueVertexOrder(Facet *facet, size_t numFacets)
  {
    size_t blockSize = 128;
    size_t numBlocks = divRoundUp(numFacets,blockSize);
    computeUniqueVertexOrderLaunch<<<(int)numBlocks,(int)blockSize>>>
      (facet,numFacets);
  }
#else
  void computeUniqueVertexOrder(Facet *facets, size_t numFacets)
  {
    parallel_for_blocked
      (0,numFacets,1024,
       [&](size_t begin, size_t end) {
         for (size_t i=begin;i<end;i++) {
           computeUniqueVertexOrder(facets[i]);
         }
       });
  }
#endif

  // ==================================================================
  // init faces
  // ==================================================================
  
  inline __umesh_both__
  void writeTetFacets(Facet *facets,
                      size_t tetIdx,
                      InputMesh mesh
                      )
  {
    for (int i=0;i<4;i++) facets[i].prim.primType = UMesh::TET;
    for (int i=0;i<4;i++) facets[i].prim.facetIdx = i;
    for (int i=0;i<4;i++) facets[i].prim.primIdx  = tetIdx;
    for (int i=0;i<4;i++) facets[i].orientation   = 0;
    
    UMesh::Tet tet = mesh.tets[tetIdx];
    facets[0].vertexIdx = { tet.y,tet.w,tet.z,-1 };
    facets[1].vertexIdx = { tet.x,tet.z,tet.w,-1 };
    facets[2].vertexIdx = { tet.x,tet.w,tet.y,-1 };
    facets[3].vertexIdx = { tet.x,tet.y,tet.z,-1 };
  }
  
  inline __umesh_both__
  void writePyrFacets(Facet *facets,
                      size_t pyrIdx,
                      InputMesh mesh
                      )
  {
    for (int i=0;i<5;i++) facets[i].prim.primType = UMesh::PYR;
    for (int i=0;i<5;i++) facets[i].prim.facetIdx = i;
    for (int i=0;i<5;i++) facets[i].prim.primIdx  = pyrIdx;
    for (int i=0;i<5;i++) facets[i].orientation   = 0;
    
    UMesh::Pyr pyr = mesh.pyrs[pyrIdx];
    vec4i base = pyr.base;
    facets[0].vertexIdx = { pyr.top,base.y,base.x,-1 };
    facets[1].vertexIdx = { pyr.top,base.z,base.y,-1 };
    facets[2].vertexIdx = { pyr.top,base.w,base.z,-1 };
    facets[3].vertexIdx = { pyr.top,base.x,base.w,-1 };
    facets[4].vertexIdx = { base.x,base.y,base.z,base.w };
  }
  
  inline __umesh_both__
  void writeWedgeFacets(Facet *facets,
                        size_t wedgeIdx,
                        InputMesh mesh
                        )
  {
    for (int i=0;i<5;i++) facets[i].prim.primType = UMesh::WEDGE;
    for (int i=0;i<5;i++) facets[i].prim.facetIdx = i;
    for (int i=0;i<5;i++) facets[i].prim.primIdx  = wedgeIdx;
    for (int i=0;i<5;i++) facets[i].orientation   = 0;
    
    UMesh::Wedge wedge = mesh.wedges[wedgeIdx];
    int i0 = wedge.front.x;
    int i1 = wedge.front.y;
    int i2 = wedge.front.z;
    int i3 = wedge.back.x;
    int i4 = wedge.back.y;
    int i5 = wedge.back.z;
    facets[0].vertexIdx = { i0,i2,i1,-1 };
    facets[1].vertexIdx = { i3,i4,i5,-1 };
    facets[2].vertexIdx = { i0,i3,i5,i2 };
    facets[3].vertexIdx = { i1,i2,i5,i4 };
    facets[4].vertexIdx = { i0,i1,i4,i3 };
  }
  
  inline __umesh_both__
  void writeHexFacets(Facet *facets,
                      size_t hexIdx,
                      InputMesh mesh
                      )
  {
    for (int i=0;i<6;i++) facets[i].prim.primType = UMesh::HEX;
    for (int i=0;i<6;i++) facets[i].prim.facetIdx = i;
    for (int i=0;i<6;i++) facets[i].prim.primIdx  = hexIdx;
    for (int i=0;i<6;i++) facets[i].orientation   = 0;
    
    UMesh::Hex hex = mesh.hexes[hexIdx];
    int i0 = hex.base.x;
    int i1 = hex.base.y;
    int i2 = hex.base.z;
    int i3 = hex.base.w;
    int i4 = hex.top.x;
    int i5 = hex.top.y;
    int i6 = hex.top.z;
    int i7 = hex.top.w;
    facets[0].vertexIdx = { i0,i1,i2,i3 };
    facets[1].vertexIdx = { i4,i7,i6,i5 };
    facets[2].vertexIdx = { i0,i4,i5,i1 };
    facets[3].vertexIdx = { i2,i6,i7,i3 };
    facets[4].vertexIdx = { i1,i5,i6,i2 };
    facets[5].vertexIdx = { i0,i3,i7,i4 };
  }
  
  inline __umesh_both__
  void writeFacets(Facet *facets, size_t jobIdx, const InputMesh &mesh)
  {
    // write tets
    if (jobIdx < mesh.numTets) {
      writeTetFacets(facets+4*jobIdx,jobIdx,mesh);
      return;
    }
    facets += 4*mesh.numTets;
    jobIdx -= mesh.numTets;
  
    // write pyramids
    if (jobIdx < mesh.numPyrs) {
      writePyrFacets(facets+5*jobIdx,jobIdx,mesh);
      return;
    }
    facets += 5*mesh.numPyrs;
    jobIdx -= mesh.numPyrs;
  
    // write wedges
    if (jobIdx < mesh.numWedges) {
      writeWedgeFacets(facets+5*jobIdx,jobIdx,mesh);
      return;
    }
    facets += 5*mesh.numWedges;
    jobIdx -= mesh.numWedges;
  
    // write hexes
    if (jobIdx < mesh.numHexes) {
      writeHexFacets(facets+6*jobIdx,jobIdx,mesh);
      return;
    }
    return;
  }

#ifdef __HIPCC__
  __global__
  void writeFacetsLaunch(Facet *facets,
                         InputMesh mesh)
  {
    size_t jobIdx = size_t(blockIdx.x)*blockDim.x+threadIdx.x;
    writeFacets(facets,jobIdx,mesh);
  }
  void writeFacets(Facet *facets,
                   const InputMesh &mesh)
  {
    size_t numPrims
      = mesh.numTets
      + mesh.numPyrs
      + mesh.numWedges
      + mesh.numHexes;
    size_t blockSize = 128;
    size_t numBlocks = divRoundUp(numPrims,blockSize);
    writeFacetsLaunch<<<(int)numBlocks,(int)blockSize>>>(facets,mesh);
  }
#else
  void writeFacets(Facet *facets,
                   const InputMesh &mesh)
  {
    size_t numPrims
      = mesh.numTets
      + mesh.numPyrs
      + mesh.numWedges
      + mesh.numHexes;
    parallel_for_blocked
      (0,numPrims,1024,
       [&](size_t begin, size_t end) {
         for (size_t i=begin;i<end;i++)
           writeFacets(facets,i,mesh);
       });
  }
#endif

  // ==================================================================
  // sort facet array
  // ==================================================================
#ifdef __HIPCC__
  void sortFacets(Facet *facets, size_t numFacets)
  {
    thrust::sort(thrust::device,
                 facets,facets+numFacets,FacetComparator());
  }
#else
  void sortFacets(Facet *facets, size_t numFacets)
  {
# ifdef UMESH_HAVE_TBB
    tbb::parallel_sort(facets,facets+numFacets,FacetComparator());
# else
    std::sort(facets,facets+numFacets,FacetComparator());
# endif
    // for (int i=0;i<100;i++)
    //   std::cout << "facet " << i << " " << facets[i].vertexIdx << std::endl;
  }
#endif
  
  // ==================================================================
  // set up / upload input mesh data
  // ==================================================================
#ifdef __HIPCC__
  template<typename T>
  inline void upload(T *&ptr, size_t &count,
                     const std::vector<T> &vec)
  {
    count = vec.size();
    if (count == 0)
      ptr = 0;
    else {
      CUDA_CALL(MallocManaged((void**)&ptr,count*sizeof(T)));
      CUDA_CALL(Memcpy(ptr,vec.data(),count*sizeof(T),hipMemcpyDefault));
    }
  }
  inline void freeInput(InputMesh &mesh)
  {
    CUDA_CALL(Free(mesh.tets));
    CUDA_CALL(Free(mesh.pyrs));
    CUDA_CALL(Free(mesh.wedges));
    CUDA_CALL(Free(mesh.hexes));
  }
  Facet *allocateFacets(size_t numFacets)
  {
    Facet *facets;
    CUDA_CALL(MallocManaged((void**)&facets,numFacets*sizeof(Facet)));
    return facets;
  }
  void freeFacets(Facet *facets)
  { CUDA_CALL(Free(facets)); }
#else
  template<typename T>
  inline void upload(T *&ptr, size_t &count,
                     const std::vector<T> &vec)
  {
    ptr = (T*)vec.data();
    count = vec.size();
  }
  inline void freeInput(InputMesh &mesh)
  {}
  Facet *allocateFacets(size_t numFacets) { return new Facet[numFacets]; }
  void freeFacets(Facet *facets)
  { delete[] facets; }
#endif

  void setupInput(InputMesh &mesh, UMesh::SP input)
  {
    upload(mesh.tets,mesh.numTets,input->tets);
    upload(mesh.pyrs,mesh.numPyrs,input->pyrs);
    upload(mesh.wedges,mesh.numWedges,input->wedges);
    upload(mesh.hexes,mesh.numHexes,input->hexes);
  }
  

  // ==================================================================
  // let facets write the facess
  // ==================================================================
  inline __umesh_both__
  void facesWriteFacesKernel(SharedFace *faces,
                             const Facet *facets,
                             const uint64_t *faceIndices,
                             size_t facetIdx)
  {
    const Facet facet = facets[facetIdx];
    size_t faceIdx = faceIndices[facetIdx]-1;
    SharedFace &face = faces[faceIdx];
    auto &side = facet.orientation ? face.onFront : face.onBack;
    face.vertexIdx = facet.vertexIdx;
    side = facet.prim;
  }
  
#ifdef __HIPCC__
  __global__ void facesWriteFacesLaunch(SharedFace *faces,
                                        const Facet *facets,
                                        const uint64_t *faceIndices,
                                        size_t numFacets)
  {
    size_t jobIdx = size_t(blockIdx.x)*blockDim.x+threadIdx.x;
    if (jobIdx >= numFacets) return;
    facesWriteFacesKernel(faces,facets,faceIndices,jobIdx);
  }
  
  void facetsWriteFaces(SharedFace *faces,
                        const Facet *facets,
                        const uint64_t *faceIndices,
                        size_t numFacets)
  {
    size_t blockSize = 128;
    size_t numBlocks = divRoundUp(numFacets,blockSize);
    facesWriteFacesLaunch<<<(int)numBlocks,(int)blockSize>>>
      (faces,facets,faceIndices,numFacets);
  }
#else
  void facetsWriteFaces(SharedFace *faces,
                        const Facet *facets,
                        const uint64_t *faceIndices,
                        size_t numFacets)
  {
    parallel_for_blocked
      (0,numFacets,1024,
       [&](size_t begin, size_t end) {
         for (size_t i=begin;i<end;i++)
           facesWriteFacesKernel(faces,facets,faceIndices,i);
       });
  }
#endif
  
  // ==================================================================
  // manage mem for faces
  // ==================================================================
#ifdef __HIPCC__
  SharedFace *allocateFaces(std::vector<SharedFace> &result,
                            size_t numFaces)
  {
    SharedFace *ptr;
    CUDA_CALL(MallocManaged((void**)&ptr,numFaces*sizeof(*ptr)));
    return ptr;
  }
  
  void finishFaces(std::vector<SharedFace> &result,
                   SharedFace *faces,
                   size_t numFaces)
  {
    result.resize(numFaces);
    CUDA_CALL(Memcpy(result.data(),faces,
                     numFaces*sizeof(*faces),hipMemcpyDefault));
    CUDA_CALL(Free(faces));
  }
#else
  SharedFace *allocateFaces(std::vector<SharedFace> &result,
                            size_t numFaces)
  {
    result.resize(numFaces);
    return result.data();
  }
  
  void finishFaces(std::vector<SharedFace> &result,
                   SharedFace *faces,
                   size_t numFaces)
  {
#if 0
    /* validate - make sure to have this off in releases */
    std::set<vec4i> knownFaces;
    for (int i=0;i<numFaces;i++) {
      vec4i idx = faces[i].vertexIdx;
      std::sort(&idx.x,&idx.x+4);
      if (knownFaces.find(idx) != knownFaces.end())
        std::cout << "validation failed: given face already exists : " << faces[i].vertexIdx << std::endl;
      knownFaces.insert(idx);
    }
    std::cout << "done validation, found " << knownFaces.size() << " unique faces" << std::endl;
#endif
    /* nothing to do */
  }
#endif

  // ==================================================================
  // manage mem for face indices
  // ==================================================================

#ifdef __HIPCC__
  void freeIndices(uint64_t *faceIndices)
  {
    CUDA_CALL(Free(faceIndices));
  }
  uint64_t *allocateIndices(size_t numFacets)
  {
    uint64_t *ptr;
    CUDA_CALL(MallocManaged((void**)&ptr,numFacets*sizeof(*ptr)));
    return ptr;
  }
#else
  void freeIndices(uint64_t *faceIndices)
  {
    delete[] faceIndices;
  }
  uint64_t *allocateIndices(size_t numFacets)
  {
    return new uint64_t[numFacets];
  }
#endif
  
  // ==================================================================
  // compute face indices from (sorted) facet array
  // ==================================================================

  inline __umesh_both__
  void initFaceIndexKernel(uint64_t *faceIndices,
                           Facet *facets,
                           size_t facetIdx)
  {
    faceIndices[facetIdx]
      =  facetIdx == 0
      || facets[facetIdx-1].vertexIdx.x != facets[facetIdx].vertexIdx.x
      || facets[facetIdx-1].vertexIdx.y != facets[facetIdx].vertexIdx.y
      || facets[facetIdx-1].vertexIdx.z != facets[facetIdx].vertexIdx.z
      || facets[facetIdx-1].vertexIdx.w != facets[facetIdx].vertexIdx.w;
  }

#ifdef __HIPCC__
  void clearFaces(SharedFace *faces, size_t numFaces)
  {
    CUDA_CALL(Memset(faces,-1,numFaces*sizeof(SharedFace)));
  }

  __global__
  void initFaceIndicesLaunch(uint64_t *faceIndices,
                             Facet *facets,
                             size_t numFacets)
  {
    size_t facetIdx = size_t(blockIdx.x)*blockDim.x+threadIdx.x;
    if (facetIdx >= numFacets) return;
    initFaceIndexKernel(faceIndices,facets,facetIdx);
  }
  
  void initFaceIndices(uint64_t *faceIndices,
                       Facet *facets,
                       size_t numFacets)
  {
    size_t blockSize = 128;
    size_t numBlocks = divRoundUp(numFacets+1,blockSize);
    initFaceIndicesLaunch<<<(int)numBlocks,(int)blockSize>>>
      (faceIndices,facets,numFacets);
  }
  void prefixSum(uint64_t *faceIndices,
                 size_t numFacets)
  {
    thrust::exclusive_scan(thrust::device,
                           faceIndices,faceIndices+numFacets,
                           faceIndices);
  }
  void postfixSum(uint64_t *faceIndices,
                 size_t numFacets)
  {
    thrust::inclusive_scan(thrust::device,
                           faceIndices,faceIndices+numFacets,
                           faceIndices);
  }
#else
  void clearFaces(SharedFace *faces, size_t numFaces)
  {
    PrimFacetRef clearPrim = { 0,0,-1 };
    for (size_t i=0;i<numFaces;i++) {
      faces[i].onFront = faces[i].onBack = clearPrim;
    }
  }
  void initFaceIndices(uint64_t *faceIndices,
                       Facet *facets,
                       size_t numFacets)
  {
    for (size_t i=0;i<numFacets;i++) {
      initFaceIndexKernel(faceIndices,facets,i);
    }
  }
  /*! not parallelized... this will likely be mem bound, anyway */
  void prefixSum(uint64_t *faceIndices,
                 size_t numFacets)
  {
    size_t sum = 0;
    for (size_t i=0;i<numFacets;i++) {
      size_t old = faceIndices[i];
      faceIndices[i] = sum;
      sum += old;
    }
  }
  /*! not parallelized... this will likely be mem bound, anyway */
  void postfixSum(uint64_t *faceIndices,
                 size_t numFacets)
  {
    size_t sum = 0;
    for (size_t i=0;i<numFacets;i++) {
      size_t old = faceIndices[i];
      sum += old;
      faceIndices[i] = sum;
    }
  }
#endif
  


  // ==================================================================
  // aaaand ... wrap it all together
  // ==================================================================

  std::vector<SharedFace> computeFaces(UMesh::SP input)
  {
    std::chrono::steady_clock::time_point
      begin_inc = std::chrono::steady_clock::now();
    InputMesh mesh;
    setupInput(mesh,input);

    std::chrono::steady_clock::time_point
      begin_exc = std::chrono::steady_clock::now();
    
    // -------------------------------------------------------
    size_t numFacets
      = 4 * mesh.numTets
      + 5 * mesh.numPyrs
      + 5 * mesh.numWedges
      + 6 * mesh.numHexes;
    Facet *facets = allocateFacets(numFacets);
    writeFacets(facets,mesh);
    computeUniqueVertexOrder(facets,numFacets);
    
    // -------------------------------------------------------
    sortFacets(facets,numFacets);
    uint64_t *faceIndices = allocateIndices(numFacets);
    initFaceIndices(faceIndices,facets,numFacets);
    postfixSum(faceIndices,numFacets);
    // prefixSum(faceIndices,numFacets);
    // -------------------------------------------------------
    size_t numFaces = faceIndices[numFacets-1]+1;
    std::vector<SharedFace> result;
    SharedFace *faces = allocateFaces(result,numFaces);
    clearFaces(faces,numFaces);
    
    // -------------------------------------------------------
    facetsWriteFaces(faces,facets,faceIndices,numFacets);
#ifdef __HIPCC__
    CUDA_SYNC_CHECK();
#endif
    std::chrono::steady_clock::time_point
      end_exc = std::chrono::steady_clock::now();
    
    finishFaces(result,faces,numFaces);
    freeIndices(faceIndices);

    std::chrono::steady_clock::time_point
      end_inc = std::chrono::steady_clock::now();
    std::cout << "done computing faces, including upload/download "
              << std::chrono::duration_cast<std::chrono::milliseconds>(end_inc - begin_inc).count()/1024.f << " secs, vs excluding "
              << std::chrono::duration_cast<std::chrono::milliseconds>(end_exc - begin_exc).count()/1024.f  << std::endl;
    return result;
  }
  
  extern "C" int main(int ac, char **av)
  {
      try {
          std::string inFileName, outFileName;
          for (int i = 1; i < ac; i++) {
              const std::string arg = av[i];
              if (arg == "-o")
                  outFileName = av[++i];
              else if (arg[0] == '-')
                  usage("unknown cmdline argument " + arg);
              else
                  inFileName = arg;
          }
          if (inFileName == "")
              throw std::runtime_error("no test file specified");
          UMesh::SP input = UMesh::loadFrom(inFileName);
          std::vector<SharedFace> result
              = computeFaces(input);
          std::cout << "done computing shared faces, found " << result.size() << " faces for mesh of " << input->toString() << std::endl;
      }
      catch (std::exception e) {
          std::cerr << "fatal error " << e.what() << std::endl;
      }
    return 0;
  } 
  
}
